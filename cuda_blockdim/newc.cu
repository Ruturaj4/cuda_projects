
#include <hip/hip_runtime.h>
#include "stdio.h"

#define N 2048*2048 //Total threads
#define THREADS_PER_BLOCK 512

__global__ void add(int *a, int *b, int *c)
{

int index = threadIdx.x + blockIdx.x * blockDim.x; //thread_no + block_no * no.ofthreads/block
c[index] = a[index] + b[index];

}

void random_ints(int* a, int n)
{

int i;
for(i =0; i<n; ++i){
a[i]=rand()/2000;
}

}

int main(void)
{

int *a,*b,*c;
int *d_a, *d_b, *d_c;
int size = N * sizeof(int);

hipMalloc((void**)&d_a, size);
hipMalloc((void**)&d_b, size);
hipMalloc((void**)&d_c, size);

a = (int*)malloc(size);
random_ints(a, N);
b = (int*)malloc(size);
random_ints(b, N);
c = (int*)malloc(size);

hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

add<<<N/THREADS_PER_BLOCK, THREADS_PER_BLOCK>>>(d_a, d_b, d_c);

hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

printf("%d + %d is %d\n", *a, *b, *c);

free(a);
free(b);
free(c);

hipFree(d_a);
hipFree(d_b);
hipFree(d_c);

return 0;

}
